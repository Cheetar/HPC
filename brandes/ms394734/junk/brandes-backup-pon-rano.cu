#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "common/errors.h"

#include <time.h>
#include <stdio.h>
#include <math.h>

#define MAX_EDGES 4000000
#define MAX_VERTICES 450000
#define THREAD_COUNT 1024
#define INF INT_MAX

__global__ void brandes_kernel(int *edges_x, int *edges_y, double *bc, double *delta, int *sigma, int *dist, int no_vertices, int no_edges) {
  int u, v, e, x;
  __shared__ bool cont;
  __shared__ int level;
  __shared__ int source;

  x = threadIdx.x;

  if (x < no_edges || x < no_vertices) {
    if (x == 0) source = -1;
    //__syncthreads();

    while (source < no_vertices - 1) {
      if (x == 0) {
        level = -1;
        cont = true;
        source++;
        //printf("\rProgress... %5.2f%%\n", (source+1)*100.0/no_vertices);
      }
      __syncthreads();
  
      for (v = x; v < no_vertices; v += blockDim.x) {
        delta[v] = 0.;
        sigma[v] = (v == source);
        dist[v] = (v == source) ? 0 : INF; 
      }
      //__syncthreads(); // TODO do wywalenia

      // Forward phase
      while (cont) {
        //__syncthreads();

        if (x == 0) level++;
        cont = false;
        __syncthreads();
        
        for (e = x; e < no_edges; e += blockDim.x) {
          u = edges_x[e];
          if (dist[u] == level) {
            v = edges_y[e];
            if (dist[v] == INF) {
              dist[v] = dist[u] + 1;
              cont = true;
            }
            if (dist[v] == (dist[u] + 1)) atomicAdd(&sigma[v], sigma[u]);
          }
        }
        __syncthreads(); 
      }
      //__syncthreads(); // TODO do wywalenia

      // Backward phase
      while (level > 0) {
        if (x == 0) level--;
        __syncthreads();

        for (e = x; e < no_edges; e += blockDim.x) {
          int u = edges_x[e];
          if (dist[u] == level) {
            int v = edges_y[e];
            if ((dist[v] == (dist[u] + 1)) && (sigma[v] != 0))
              atomicAdd(&delta[u], (sigma[u] * 1.0 / sigma[v]) * (1 + delta[v]));
          }
        }
        __syncthreads();
      }

      for (v = x; v < no_vertices; v += blockDim.x)
          if (v != source) bc[v] += delta[v];
      //__syncthreads();
    }
  }
}

int main(int argc, char **argv) {
  if (argc != 3) {
    printf("usage: ./brandes <input-file> <output-file>\n");
    exit(1);
  }

  FILE *in_file;
  FILE *out_file;
  in_file = fopen(argv[1], "r");
  out_file = fopen(argv[2], "w+");

  // Host data initialization
  int a, b, no_edges = 0, no_vertices = 0;
  int* edges_x = (int*)malloc(MAX_EDGES * sizeof(int));
  int* edges_y = (int*)malloc(MAX_EDGES * sizeof(int));
 
  fscanf (in_file, "%d %d", &a, &b);
  edges_x[no_edges] = a;
  edges_y[no_edges] = b;
  no_edges += 1;
  edges_x[no_edges] = b;
  edges_y[no_edges] = a;
  no_edges += 1;
  no_vertices = max(max(a, b), no_vertices - 1) + 1;

  while (!feof (in_file)) {
      fscanf(in_file, "%d %d\n", &a, &b);    
      edges_x[no_edges] = a;
      edges_y[no_edges] = b;  
      no_edges += 1;
      edges_x[no_edges] = b;
      edges_y[no_edges] = a;  
      no_edges += 1;
      no_vertices = max(max(a, b), no_vertices - 1) + 1;

      if (no_edges > MAX_EDGES) {
        printf("More edges than allowed (2 mln)\n");
        exit(1);
      }
  }

  double *bc = (double*)malloc(no_vertices*sizeof(double));

  // CUDA malloc
  int *dist, *sigma, *dev_edges_x, *dev_edges_y; 
  double *dev_bc, *delta;
  
  HANDLE_ERROR(hipMalloc((int**)&dev_edges_x, no_edges * sizeof(int)));
  HANDLE_ERROR(hipMalloc((int**)&dev_edges_y, no_edges * sizeof(int)));
  HANDLE_ERROR(hipMalloc((int**)&dist, no_vertices * sizeof(int)));
  HANDLE_ERROR(hipMalloc((int**)&sigma, no_vertices * sizeof(int)));
  HANDLE_ERROR(hipMalloc((double**)&dev_bc, no_vertices * sizeof(double)));
  HANDLE_ERROR(hipMalloc((double**)&delta, no_vertices * sizeof(double)));

  HANDLE_ERROR(hipMemset(dev_bc, 0, no_vertices * sizeof(double)));


  // Transfer data to the device
  hipEvent_t start, start_kernel, stop, stop_kernel;
  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&stop));
  HANDLE_ERROR(hipEventRecord(start, 0));
    
  HANDLE_ERROR(hipMemcpy(dev_edges_x, edges_x, no_edges * sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_edges_y, edges_y, no_edges * sizeof(int), hipMemcpyHostToDevice));


  // Run the kernel
  HANDLE_ERROR(hipEventCreate(&start_kernel));
  HANDLE_ERROR(hipEventCreate(&stop_kernel));
  HANDLE_ERROR(hipEventRecord(start_kernel, 0));

  brandes_kernel<<<1, THREAD_COUNT>>>(dev_edges_x, dev_edges_y, dev_bc, delta, sigma, dist, no_vertices, no_edges);

  HANDLE_ERROR(hipEventRecord(stop_kernel, 0));
  HANDLE_ERROR(hipEventSynchronize(stop_kernel));
  float elapsed_time_kernel;
  HANDLE_ERROR(hipEventElapsedTime(&elapsed_time_kernel, start_kernel, stop_kernel));
  printf("Kernel running time: %3.1f ms\n", elapsed_time_kernel);
  HANDLE_ERROR(hipEventDestroy(start_kernel));
  HANDLE_ERROR(hipEventDestroy(stop_kernel));
  

  // Transfer results from device to the host
  HANDLE_ERROR(hipMemcpy(bc, dev_bc, no_vertices*sizeof(double), hipMemcpyDeviceToHost));

  HANDLE_ERROR(hipEventRecord(stop, 0));
  HANDLE_ERROR(hipEventSynchronize(stop));
  float elapsedTime;
  HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
  printf("Total time: %3.1f ms\n", elapsedTime);
  HANDLE_ERROR(hipEventDestroy(start));
  HANDLE_ERROR(hipEventDestroy(stop));


  // Save the results to the output file
  for (int i = 0; i < no_vertices; i++) {
    fprintf(out_file, "%f\n", bc[i]);
  }

  // Clean up 
  free(edges_x);
  free(edges_y);
  free(bc);
  hipFree(dev_edges_x);
  hipFree(dev_edges_y);
  hipFree(dev_bc);
  hipFree(delta);
  hipFree(sigma);
  hipFree(dist);

  fclose(in_file);
  fclose(out_file);

  return 0;
}