#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "common/errors.h"

#include <time.h>
#include <stdio.h>
#include <math.h>

#define MAX_EDGES 4000000
#define MAX_VERTICES 450000
#define THREADS_PER_BLOCK 128
#define BLOCKS 1024
#define INF INT_MAX


__device__ __managed__ bool dev_cont;


__global__ void initialize(int *sigma, int *dist, double *delta, int no_vertices, int source) {
  
  int v;
  int idx = blockDim.x * blockIdx.x + threadIdx.x;

  for (v = idx; v < no_vertices; v += gridDim.x * blockDim.x) {
    sigma[v] = 0;
    dist[v] = INF;
    delta[v] = 0.;
  }

  if (idx == 0) {
    sigma[source] = 1;
    dist[source] = 0;
  }
}

__global__ void forward(int *edges_x, int *edges_y, int *sigma, int *dist, int no_edges, int level) {

  int e, u, v;
  int idx = blockDim.x * blockIdx.x + threadIdx.x;

  for (e = idx; e < no_edges; e += gridDim.x * blockDim.x) {
    u = edges_x[e];
    if (dist[u] == level) {
      v = edges_y[e];
      if (dist[v] == INF) {
        dist[v] = level + 1;
        dev_cont = true;
      }
      if (dist[v] == (level + 1)) atomicAdd(&sigma[v], sigma[u]);
    }
  }
}

__global__ void backward(int *edges_x, int *edges_y, double *delta, int *sigma, int *dist, int no_edges, int level) {

  int e, u, v;
  int idx = blockDim.x * blockIdx.x + threadIdx.x;

  for (e = idx; e < no_edges; e += gridDim.x * blockDim.x) {
    u = edges_x[e];
    if (dist[u] == level) {
      v = edges_y[e];
      if ((dist[v] == (dist[u] + 1)) && (sigma[v] != 0))
        atomicAdd(&delta[u], (sigma[u] * 1.0 / sigma[v]) * (1 + delta[v]));
    }
  }
}

__global__ void update_bc(double *bc, double *delta, int no_vertices, int source) {

  int v;
  int idx = blockDim.x * blockIdx.x + threadIdx.x;

  for (v = idx; v < no_vertices; v += gridDim.x * blockDim.x)
    //if (v != source)
    //bc[v] += delta[v] * (int)(v != source);
    atomicAdd(&bc[v], delta[v] * (int)(v != source));
}

int main(int argc, char **argv) {
  hipFree(0);

  if (argc != 3) {
    printf("usage: ./brandes <input-file> <output-file>\n");
    exit(1);
  }

  FILE *in_file;
  FILE *out_file;
  in_file = fopen(argv[1], "r");
  out_file = fopen(argv[2], "w+");

  // Host data initialization
  int a, b, no_edges = 0, no_vertices = 0;
  int* edges_x = (int*)malloc(MAX_EDGES * sizeof(int));
  int* edges_y = (int*)malloc(MAX_EDGES * sizeof(int));
  int* deg = (int*)malloc(MAX_VERTICES * sizeof(int));
  memset(deg, 0, MAX_VERTICES * sizeof(int));

  fscanf (in_file, "%d %d", &a, &b);
  edges_x[no_edges] = a;
  edges_y[no_edges] = b;
  no_edges += 1;
  edges_x[no_edges] = b;
  edges_y[no_edges] = a;
  no_edges += 1;
  deg[a]++;
  deg[b]++;
  no_vertices = max(max(a, b), no_vertices - 1) + 1;

  while (!feof (in_file)) {
    fscanf(in_file, "%d %d\n", &a, &b);    
    edges_x[no_edges] = a;
    edges_y[no_edges] = b;  
    no_edges += 1;
    edges_x[no_edges] = b;
    edges_y[no_edges] = a;  
    no_edges += 1;
    deg[a]++;
    deg[b]++;
    no_vertices = max(max(a, b), no_vertices - 1) + 1;

    if (no_edges > MAX_EDGES) {
      printf("More edges than allowed (2 mln)\n");
      exit(1);
    }
  }

  printf("no_vertices %d\n", no_vertices);
  printf("no_edges %d\n", no_edges);

  /*
  int no_deg[200] = {0}; 

  for (int i = 0; i < no_vertices; i++) {
    if (deg[i] < 199)
      no_deg[deg[i]]++;
    else
     no_deg[199]++;
  }

  for (int i = 0; i < 200; i++) {
    printf("no_deg[%d] = %d\n", i, no_deg[i]);
  }
*/
  double *bc = (double*)malloc(no_vertices * sizeof(double));

  // CUDA malloc
  int *dist, *sigma, *dev_edges_x, *dev_edges_y; 
  double *dev_bc, *delta;
  
  HANDLE_ERROR(hipMalloc((int**)&dev_edges_x, no_edges * sizeof(int)));
  HANDLE_ERROR(hipMalloc((int**)&dev_edges_y, no_edges * sizeof(int)));

  HANDLE_ERROR(hipMalloc((int**)&dist, no_vertices * sizeof(int)));
  HANDLE_ERROR(hipMalloc((int**)&sigma, no_vertices * sizeof(int)));
  HANDLE_ERROR(hipMalloc((double**)&dev_bc, no_vertices * sizeof(double)));
  HANDLE_ERROR(hipMalloc((double**)&delta, no_vertices* sizeof(double)));

  HANDLE_ERROR(hipMemset(dev_bc, 0, no_vertices * sizeof(double)));


  // Transfer data to the device
  hipEvent_t start, start_kernel, stop, stop_kernel;
  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&stop));
  HANDLE_ERROR(hipEventRecord(start, 0));
    
  HANDLE_ERROR(hipMemcpy(dev_edges_x, edges_x, no_edges * sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_edges_y, edges_y, no_edges * sizeof(int), hipMemcpyHostToDevice));


  // Run the kernel
  HANDLE_ERROR(hipEventCreate(&start_kernel));
  HANDLE_ERROR(hipEventCreate(&stop_kernel));
  HANDLE_ERROR(hipEventRecord(start_kernel, 0));

  // brandes_kernel<<<BLOCKS, THREADS_PER_BLOCK>>>(dev_edges_x, dev_edges_y, dev_bc, delta, sigma, dist, no_vertices, no_edges);
  int source, level;
  //bool *cont;
  //cont = (bool*)malloc(sizeof(bool));

  //HANDLE_ERROR(hipHostMalloc((void**)&cont, sizeof(bool))); // Allocate pinned memory
  //hipHostAlloc((void**)&cont, sizeof(bool), hipHostMallocDefault);
  //HANDLE_ERROR(hipMalloc((void**)&dev_cont, sizeof(bool)));


  for (source = 0; source < no_vertices; source++) {
    if (source % 1000 == 0)
      printf("Progress (%d/%d) %.2f%\n", source, no_vertices, source * 100.0 / no_vertices);

    // Initialization
    initialize<<<BLOCKS,THREADS_PER_BLOCK>>>(sigma, dist, delta, no_vertices, source);
    hipDeviceSynchronize();

    // Forward phase
    level = 0;
    //*cont = true;
    dev_cont = true;
    //while (*cont) {
    while (dev_cont) {
      // Update cont and send current value to device
      //*cont = false;
      dev_cont = false;
      //HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(dev_cont), cont, sizeof(bool)));
      //hipDeviceSynchronize();

      forward<<<BLOCKS,THREADS_PER_BLOCK>>>(dev_edges_x, dev_edges_y, sigma, dist, no_edges, level);
      //hipDeviceSynchronize();

      level++;
      // Load cont value from device
      //HANDLE_ERROR(hipMemcpyFromSymbol(cont, HIP_SYMBOL(dev_cont), sizeof(bool)));
      //hipDeviceSynchronize();
    }

    // Backward phase
    while (level > 0) { // was 1 
      level--;
      backward<<<BLOCKS,THREADS_PER_BLOCK>>>(dev_edges_x, dev_edges_y, delta, sigma, dist, no_edges, level);
      hipDeviceSynchronize();
    }

    update_bc<<<BLOCKS,THREADS_PER_BLOCK>>>(dev_bc, delta, no_vertices, source);
    hipDeviceSynchronize();
  }

  //HANDLE_ERROR(hipDeviceSynchronize());
  HANDLE_ERROR(hipEventRecord(stop_kernel, 0));
  HANDLE_ERROR(hipEventSynchronize(stop_kernel));
  float elapsed_time_kernel;
  HANDLE_ERROR(hipEventElapsedTime(&elapsed_time_kernel, start_kernel, stop_kernel));
  printf("Kernel running time: %3.1f ms\n", elapsed_time_kernel);
  HANDLE_ERROR(hipEventDestroy(start_kernel));
  HANDLE_ERROR(hipEventDestroy(stop_kernel));
  

  // Transfer results from device to the host
  HANDLE_ERROR(hipMemcpy(bc, dev_bc, no_vertices * sizeof(double), hipMemcpyDeviceToHost));

  HANDLE_ERROR(hipEventRecord(stop, 0));
  HANDLE_ERROR(hipEventSynchronize(stop));
  float elapsedTime;
  HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
  printf("Total time: %3.1f ms\n", elapsedTime);
  HANDLE_ERROR(hipEventDestroy(start));
  HANDLE_ERROR(hipEventDestroy(stop));


  // Save the results to the output file
  for (int i = 0; i < no_vertices; i++) {
    fprintf(out_file, "%f\n", bc[i]);
  }

  // Clean up 
  free(edges_x);
  free(edges_y);
  free(bc);
  //hipFree(cont);
  //hipFree(dev_cont);
  hipFree(dev_edges_x);
  hipFree(dev_edges_y);
  hipFree(dev_bc);
  hipFree(delta);
  hipFree(sigma);
  hipFree(dist);

  fclose(in_file);
  fclose(out_file);

  return 0;
}