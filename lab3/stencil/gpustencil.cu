#include "hip/hip_runtime.h"
#include <time.h>
#include <stdio.h>
#include <math.h>

#define RADIUS        3000
#define NUM_ELEMENTS  1000000

static void handleError(hipError_t err, const char *file, int line ) {
  if (err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
  }
}
#define cudaCheck( err ) (handleError(err, __FILE__, __LINE__ ))

__global__ void stencil_1d(int *in, int *out) {
  //PUT YOUR CODE HERE
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < NUM_ELEMENTS) {
    int total = 0;
    for (int j = max(0, i - RADIUS); j < min(NUM_ELEMENTS, i + RADIUS); j++) {
      total += in[j];
    }
    out[i] = total;
  }
}

void cpu_stencil_1d(int *in, int *out) {
  for (int i = 0; i < NUM_ELEMENTS; i++) {
    int total = 0;
    for (int j = max(0, i - RADIUS); j < min(NUM_ELEMENTS, i + RADIUS); j++) {
      total += in[j];
    }
    out[i] = total;
  }
}

int main() {
  //PUT YOUR CODE HERE - INPUT AND OUTPUT ARRAYS
  int *in, *out, *d_in, *d_out;

  in  = (int*)malloc(sizeof(int) * NUM_ELEMENTS);
  out = (int*)malloc(sizeof(int) * NUM_ELEMENTS);

  for (int i = 0; i < NUM_ELEMENTS; i++) {
    in[i]  = 1;
    out[i] = 0;
  }

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord( start, 0 );

  //PUT YOUR CODE HERE - DEVICE MEMORY ALLOCATION
  cudaCheck(hipMalloc((void**)&d_in,  sizeof(int) * NUM_ELEMENTS));
  cudaCheck(hipMalloc((void**)&d_out, sizeof(int) * NUM_ELEMENTS));

  cudaCheck(hipMemcpy(d_in,  in, sizeof(int) * NUM_ELEMENTS, hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(d_out, in, sizeof(int) * NUM_ELEMENTS, hipMemcpyHostToDevice));

  hipEvent_t start_k, stop_k;
  hipEventCreate(&start_k);
  hipEventCreate(&stop_k);
  hipEventRecord( start_k, 0 );

  //PUT YOUR CODE HERE - KERNEL EXECUTION
  stencil_1d<<<((NUM_ELEMENTS+1024)/1024), 1024>>>(d_in, d_out);

  hipEventRecord(stop_k, 0);
  hipEventSynchronize(stop_k);
  float elapsedTime_k;
  hipEventElapsedTime( &elapsedTime_k, start_k, stop_k);
  printf("GPU kernel execution time:  %3.1f ms\n", elapsedTime_k);
  hipEventDestroy(start_k);
  hipEventDestroy(stop_k);

  cudaCheck(hipPeekAtLastError());

  //PUT YOUR CODE HERE - COPY RESULT FROM DEVICE TO HOST
  cudaCheck(hipMemcpy(out, d_out, sizeof(int) * NUM_ELEMENTS, hipMemcpyDeviceToHost));
  
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float elapsedTime;
  hipEventElapsedTime( &elapsedTime, start, stop);
  printf("Total GPU execution time:  %3.1f ms\n", elapsedTime);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  //PUT YOUR CODE HERE - FREE DEVICE MEMORY  
  cudaCheck(hipFree(d_in));
  cudaCheck(hipFree(d_out));

  for (int i = 0; i < NUM_ELEMENTS; i++) {
    in[i]  = 1;
    out[i] = 0;
  }

  struct timespec cpu_start, cpu_stop;
  clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &cpu_start);

  cpu_stencil_1d(in, out);

  clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &cpu_stop);
  double result = (cpu_stop.tv_sec - cpu_start.tv_sec) * 1e3 + (cpu_stop.tv_nsec - cpu_start.tv_nsec) / 1e6;
  printf( "CPU execution time:  %3.1f ms\n", result);

  return 0;
}


